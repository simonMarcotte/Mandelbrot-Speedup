#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>

#define CUDA_CHECK(call) \
do { \
    hipError_t result_ = (call); \
    if (result_ != hipSuccess) { \
        fprintf(stderr, "%s:%d: CUDA error %d: %s\n", __FILE__, __LINE__, result_, hipGetErrorString(result_)); \
        exit(1); \
    } \
} while (0)

__device__ int mandelbrot(double x, double y, int maxiter) {
    double u = 0.0;
    double v = 0.0;
    double u2 = u * u;
    double v2 = v * v;
    int k;
    for (k = 1; k < maxiter && (u2 + v2 < 4.0); k++) {
        v = 2 * u * v + y;
        u = u2 - v2 + x;
        u2 = u * u;
        v2 = v * v;
    }
    return k;
}

__global__ void mandelbrotKernel(double xmin, double ymin, double dx, double dy, int maxiter, int xres, int yres, unsigned char* result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < xres && j < yres) {
        double x = xmin + i * dx;
        double y = ymin + j * dy;
        int k = mandelbrot(x, y, maxiter);
        int index = (j * xres + i) * 6;
        if (k >= maxiter) {
            // Interior
            result[index] = 0;
            result[index + 1] = 0;
            result[index + 2] = 0;
            result[index + 3] = 0;
            result[index + 4] = 0;
            result[index + 5] = 0;
        } else {
            // Exterior
            result[index] = k >> 8;
            result[index + 1] = k & 255;
            result[index + 2] = k >> 8;
            result[index + 3] = k & 255;
            result[index + 4] = k >> 8;
            result[index + 5] = k & 255;
        }
    }
}

int main(int argc, char* argv[]) {
    if (argc != 8) {
        printf("Usage:   %s <xmin> <xmax> <ymin> <ymax> <maxiter> <xres> <out.ppm>\n", argv[0]);
        printf("test");
        printf("Example: %s 0.27085 0.27100 0.004640 0.004810 1000 1024 pic.ppm\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    time_t start, end;

    time(&start);

    const double xmin = atof(argv[1]);
    const double xmax = atof(argv[2]);
    const double ymin = atof(argv[3]);
    const double ymax = atof(argv[4]);
    const uint16_t maxiter = (unsigned short)atoi(argv[5]);
    const int xres = atoi(argv[6]);
    const int yres = (xres * (ymax - ymin)) / (xmax - xmin);
    const char* filename = argv[7];

    FILE* fp = fopen(filename, "wb");
    char* comment = "# Mandelbrot set";
    fprintf(fp,
            "P6\n# Mandelbrot, xmin=%lf, xmax=%lf, ymin=%lf, ymax=%lf, maxiter=%d\n%d\n%d\n%d\n",
            xmin, xmax, ymin, ymax, maxiter, xres, yres, (maxiter < 256 ? 256 : maxiter));

    double dx = (xmax - xmin) / xres;
    double dy = (ymax - ymin) / yres;

    unsigned char* result;
    CUDA_CHECK(hipMallocManaged(&result, xres * yres * 6 * sizeof(unsigned char)));

    dim3 blockSize(16, 16);
    dim3 gridSize((xres + blockSize.x - 1) / blockSize.x, (yres + blockSize.y - 1) / blockSize.y);
    mandelbrotKernel<<<gridSize, blockSize>>>(xmin, ymin, dx, dy, maxiter, xres, yres, result);
    CUDA_CHECK(hipDeviceSynchronize());

    fwrite(result, xres * yres * 6, 1, fp);
    fclose(fp);
    hipFree(result);

    time(&end);
    double time_taken = double(end - start);
    
    printf("Generating Cuda Mandelbrot took: %.5f s", time_taken);

    return 0;
}
